#include "hip/hip_runtime.h"
#include "tracers.h"

#include "platform.h"

#include "libgi/timer.h"

#include <iostream>

#include "kernels.h"
#include "cuda-helpers.h"

namespace wf {
	namespace cuda {

		void simple_rt::build(::scene *scene) {
			rd = new raydata(rc->resolution());
			sd = new scenedata;

			binary_bvh_tracer<bbvh_triangle_layout::indexed, bbvh_esc_mode::on> bvh_rt;
			if (bvh_type == "sah") bvh_rt.binary_split_type = bvh_rt.sah;
			else if (bvh_type == "sm") bvh_rt.binary_split_type = bvh_rt.sm;
			else if (bvh_type == "om") bvh_rt.binary_split_type = bvh_rt.om;
			bvh_rt.max_triangles_per_node = bvh_max_tris_per_node;
			bvh_rt.build(scene);

			std::vector<wf::cuda::simple_bvh_node> nodes;
			for (const auto &n : bvh_rt.nodes) {
				wf::cuda::simple_bvh_node node(n);
				nodes.push_back(node);
			}
			assert(nodes.size() == bvh_rt.nodes.size());
			bvh_index.upload(bvh_rt.index);
			bvh_nodes.upload(nodes);

			sd->upload(scene);
			std::cout << "upload done" << std::endl;
		}

		void simple_rt::compute_hit(bool anyhit) {
			int2 resolution{rc->resolution().x, rc->resolution().y};
			simple_trace<<<NUM_BLOCKS_FOR_RESOLUTION(resolution), DESIRED_BLOCK_SIZE>>>(resolution,
																						rd->rays.device_memory,
																						sd->vertex_pos.device_memory,
																						sd->triangles.device_memory,
																						bvh_index.device_memory,
																						bvh_nodes.device_memory,
																						rd->intersections.device_memory,
																						anyhit);
			CHECK_CUDA_ERROR(hipGetLastError(), "");
  			CHECK_CUDA_ERROR(hipDeviceSynchronize(), "");
		}


		void ifif::compute_hit(bool anyhit) {
			int2 resolution{rc->resolution().x, rc->resolution().y};
			ifif_trace<<<NUM_BLOCKS_FOR_RESOLUTION(resolution), DESIRED_BLOCK_SIZE>>>(resolution,
																					  rd->rays.device_memory,       rd->rays.tex,
																					  sd->vertex_pos.device_memory, sd->vertex_pos.tex,
																					  sd->triangles.device_memory,  sd->triangles.tex,
																					  bvh_index.device_memory,      bvh_index.tex,
																					  bvh_nodes.device_memory,      bvh_nodes.tex,
																					  rd->intersections.device_memory,
																					  anyhit);
			CHECK_CUDA_ERROR(hipGetLastError(), "");
  			CHECK_CUDA_ERROR(hipDeviceSynchronize(), "");
		}
		
		
		void whilewhile::compute_hit(bool anyhit) {
			int2 resolution{rc->resolution().x, rc->resolution().y};
			whilewhile_trace<<<NUM_BLOCKS_FOR_RESOLUTION(resolution), DESIRED_BLOCK_SIZE>>>(resolution,
																							rd->rays.device_memory,       rd->rays.tex,
																							sd->vertex_pos.device_memory, sd->vertex_pos.tex,
																							sd->triangles.device_memory,  sd->triangles.tex,
																							pf->rt->bvh_index.device_memory,  pf->rt->bvh_index.tex,
																							pf->rt->bvh_nodes.device_memory,  pf->rt->bvh_nodes.tex,
																							rd->intersections.device_memory,
																							anyhit);
			CHECK_CUDA_ERROR(hipGetLastError(), "");
  			CHECK_CUDA_ERROR(hipDeviceSynchronize(), "");
		}
		
		
		void dynamicwhilewhile::compute_hit(bool anyhit) {
			// dynamicwhilewhile_trace<<<DESIRED_BLOCKS_COUNT, DESIRED_BLOCK_SIZE>>>(
			// dynamicwhilewhile-Kernel uses 48 Registers instead of 40, so run one less warp than usual for best occupation
			dynamicwhilewhile_trace<<<DESIRED_BLOCKS_COUNT, dim3(WARPSIZE, DESIRED_WARPS_PER_BLOCK-1, 1)>>>(rc->resolution().x * rc->resolution().y,
																											rd->rays.device_memory,       rd->rays.tex,
																											sd->vertex_pos.device_memory, sd->vertex_pos.tex,
																											sd->triangles.device_memory,  sd->triangles.tex,
																											pf->rt->bvh_index.device_memory,  pf->rt->bvh_index.tex,
																											pf->rt->bvh_nodes.device_memory,  pf->rt->bvh_nodes.tex,
																											rd->intersections.device_memory,
																											anyhit);
			CHECK_CUDA_ERROR(hipGetLastError(), "");
  			CHECK_CUDA_ERROR(hipDeviceSynchronize(), "");
		}
		
		
		void speculativewhilewhile::compute_hit(bool anyhit) {
			int2 resolution{rc->resolution().x, rc->resolution().y};
			speculativewhilewhile_trace<<<NUM_BLOCKS_FOR_RESOLUTION(resolution), DESIRED_BLOCK_SIZE>>>(resolution,
																									   rd->rays.device_memory,       rd->rays.tex,
																									   sd->vertex_pos.device_memory, sd->vertex_pos.tex,
																									   sd->triangles.device_memory,  sd->triangles.tex,
																									   pf->rt->bvh_index.device_memory,  pf->rt->bvh_index.tex,
																									   pf->rt->bvh_nodes.device_memory,  pf->rt->bvh_nodes.tex,
																									   rd->intersections.device_memory,
																									   anyhit);
			CHECK_CUDA_ERROR(hipGetLastError(), "");
  			CHECK_CUDA_ERROR(hipDeviceSynchronize(), "");
		}
		
		
		void persistentifif::compute_hit(bool anyhit) {
			persistentifif_trace<<<DESIRED_BLOCKS_COUNT, DESIRED_BLOCK_SIZE>>>(rc->resolution().x*rc->resolution().y,
																			   rd->rays.device_memory,       rd->rays.tex,
																			   sd->vertex_pos.device_memory, sd->vertex_pos.tex,
																			   sd->triangles.device_memory,  sd->triangles.tex,
																			   pf->rt->bvh_index.device_memory,  pf->rt->bvh_index.tex,
																			   pf->rt->bvh_nodes.device_memory,  pf->rt->bvh_nodes.tex,
																			   rd->intersections.device_memory,
																			   anyhit);
			CHECK_CUDA_ERROR(hipGetLastError(), "");
  			CHECK_CUDA_ERROR(hipDeviceSynchronize(), "");
		}
		
		
		void persistentspeculativewhilewhile::compute_hit(bool anyhit) {
			persistentspeculativewhilewhile_trace<<<DESIRED_BLOCKS_COUNT, DESIRED_BLOCK_SIZE>>>(rc->resolution().x * rc->resolution().y,
																								rd->rays.device_memory,       rd->rays.tex,
																								sd->vertex_pos.device_memory, sd->vertex_pos.tex,
																								sd->triangles.device_memory,  sd->triangles.tex,
																								pf->rt->bvh_index.device_memory,  pf->rt->bvh_index.tex,
																								pf->rt->bvh_nodes.device_memory,  pf->rt->bvh_nodes.tex,
																								rd->intersections.device_memory,
																								anyhit);
			CHECK_CUDA_ERROR(hipGetLastError(), "");
  			CHECK_CUDA_ERROR(hipDeviceSynchronize(), "");
		}
		
		
		void persistentwhilewhile::compute_hit(bool anyhit) {
			persistentwhilewhile_trace<<<DESIRED_BLOCKS_COUNT, DESIRED_BLOCK_SIZE>>>(rc->resolution().x*rc->resolution().y,
																					 rd->rays.device_memory,       rd->rays.tex,
																					 sd->vertex_pos.device_memory, sd->vertex_pos.tex,
																					 sd->triangles.device_memory,  sd->triangles.tex,
																					 pf->rt->bvh_index.device_memory,  pf->rt->bvh_index.tex,
																					 pf->rt->bvh_nodes.device_memory,  pf->rt->bvh_nodes.tex,
																					 rd->intersections.device_memory,
																					 anyhit);
			CHECK_CUDA_ERROR(hipGetLastError(), "");
  			CHECK_CUDA_ERROR(hipDeviceSynchronize(), "");
		}

	}
}
