#include "hip/hip_runtime.h"
#include "tracers.h"

#include "libgi/timer.h"

#include <iostream>

#include "kernels.h"
#include "cuda-helpers.h"

namespace wf {
	namespace cuda {

		void simple_rt::build(::scene *scene) {
			rd = new raydata(rc->resolution());
			sd = new scenedata;

			binary_bvh_tracer<bbvh_triangle_layout::indexed, bbvh_esc_mode::on> bvh_rt;
			if (bvh_type == "sah") bvh_rt.binary_split_type = bvh_rt.sah;
			else if (bvh_type == "sm") bvh_rt.binary_split_type = bvh_rt.sm;
			else if (bvh_type == "om") bvh_rt.binary_split_type = bvh_rt.om;
			bvh_rt.max_triangles_per_node = bvh_max_tris_per_node;
			bvh_rt.build(scene);

			std::vector<wf::cuda::simple_bvh_node> nodes;
			for (const auto &n : bvh_rt.nodes) {
				wf::cuda::simple_bvh_node node(n);
				nodes.push_back(node);
			}
			assert(nodes.size() == bvh_rt.nodes.size());
			bvh_index.upload(bvh_rt.index);
			bvh_nodes.upload(nodes);

			auto *rt = dynamic_cast<batch_rt*>(rc->scene.batch_rt);
			assert(rt != nullptr);
			sd->upload(scene);
			std::cout << "upload done" << std::endl;
		}

		void simple_rt::compute_hit(bool anyhit) {
			auto *rt = dynamic_cast<wf::cuda::simple_rt*>(rc->scene.batch_rt);
			assert(rt != nullptr);
			int2 resolution{rc->resolution().x, rc->resolution().y};

			simple_trace<<<NUM_BLOCKS_FOR_RESOLUTION(resolution), DESIRED_BLOCK_SIZE>>>(resolution,
																						rd->rays.device_memory,
																						sd->vertex_pos.device_memory,
																						sd->triangles.device_memory,
																						bvh_index.device_memory,
																						bvh_nodes.device_memory,
																						rd->intersections.device_memory,
																						anyhit);
			CHECK_CUDA_ERROR(hipGetLastError());
  			CHECK_CUDA_ERROR(hipDeviceSynchronize());
		}


		void ifif::compute_hit(bool anyhit) {
			auto *rt = dynamic_cast<wf::cuda::ifif*>(rc->scene.batch_rt);
			assert(rt != nullptr);
			int2 resolution{rc->resolution().x, rc->resolution().y};

			ifif_trace<<<NUM_BLOCKS_FOR_RESOLUTION(resolution), DESIRED_BLOCK_SIZE>>>(resolution,
																					  rd->rays.device_memory,       rd->rays.tex,
																					  sd->vertex_pos.device_memory, sd->vertex_pos.tex,
																					  sd->triangles.device_memory,  sd->triangles.tex,
																					  bvh_index.device_memory,      bvh_index.tex,
																					  bvh_nodes.device_memory,      bvh_nodes.tex,
																					  rd->intersections.device_memory,
																					  anyhit);
			CHECK_CUDA_ERROR(hipGetLastError());
  			CHECK_CUDA_ERROR(hipDeviceSynchronize());
		}
		
		
		void whilewhile::compute_hit(bool anyhit) {
			auto *rt = dynamic_cast<wf::cuda::whilewhile*>(rc->scene.batch_rt);
			assert(rt != nullptr);
			int2 resolution{rc->resolution().x, rc->resolution().y};
			whilewhile_trace<<<NUM_BLOCKS_FOR_RESOLUTION(resolution), DESIRED_BLOCK_SIZE>>>(resolution,
																							rd->rays.device_memory,       rd->rays.tex,
																							sd->vertex_pos.device_memory, sd->vertex_pos.tex,
																							sd->triangles.device_memory,  sd->triangles.tex,
																							rt->bvh_index.device_memory,  rt->bvh_index.tex,
																							rt->bvh_nodes.device_memory,  rt->bvh_nodes.tex,
																							rd->intersections.device_memory,
																							anyhit);
			CHECK_CUDA_ERROR(hipGetLastError());
  			CHECK_CUDA_ERROR(hipDeviceSynchronize());
		}
		
		
		void dynamicwhilewhile::compute_hit(bool anyhit) {
			auto *rt = dynamic_cast<wf::cuda::dynamicwhilewhile*>(rc->scene.batch_rt);
			assert(rt != nullptr);

			// dynamicwhilewhile_trace<<<DESIRED_BLOCKS_COUNT, DESIRED_BLOCK_SIZE>>>(
			// dynamicwhilewhile-Kernel uses 48 Registers instead of 40, so run one less warp than usual for best occupation
			dynamicwhilewhile_trace<<<DESIRED_BLOCKS_COUNT, dim3(WARPSIZE, DESIRED_WARPS_PER_BLOCK-1, 1)>>>(rc->resolution().x * rc->resolution().y,
																											rd->rays.device_memory,       rd->rays.tex,
																											sd->vertex_pos.device_memory, sd->vertex_pos.tex,
																											sd->triangles.device_memory,  sd->triangles.tex,
																											rt->bvh_index.device_memory,  rt->bvh_index.tex,
																											rt->bvh_nodes.device_memory,  rt->bvh_nodes.tex,
																											rd->intersections.device_memory,
																											anyhit);
			CHECK_CUDA_ERROR(hipGetLastError());
  			CHECK_CUDA_ERROR(hipDeviceSynchronize());
		}
		
		
		void speculativewhilewhile::compute_hit(bool anyhit) {
			auto *rt = dynamic_cast<wf::cuda::speculativewhilewhile*>(rc->scene.batch_rt);
			assert(rt != nullptr);
			int2 resolution{rc->resolution().x, rc->resolution().y};

			speculativewhilewhile_trace<<<NUM_BLOCKS_FOR_RESOLUTION(resolution), DESIRED_BLOCK_SIZE>>>(resolution,
																									   rd->rays.device_memory,       rd->rays.tex,
																									   sd->vertex_pos.device_memory, sd->vertex_pos.tex,
																									   sd->triangles.device_memory,  sd->triangles.tex,
																									   rt->bvh_index.device_memory,  rt->bvh_index.tex,
																									   rt->bvh_nodes.device_memory,  rt->bvh_nodes.tex,
																									   rd->intersections.device_memory,
																									   anyhit);
			CHECK_CUDA_ERROR(hipGetLastError());
  			CHECK_CUDA_ERROR(hipDeviceSynchronize());
		}
		
		
		void persistentifif::compute_hit(bool anyhit) {
			auto *rt = dynamic_cast<wf::cuda::persistentifif*>(rc->scene.batch_rt);
			assert(rt != nullptr);

			persistentifif_trace<<<DESIRED_BLOCKS_COUNT, DESIRED_BLOCK_SIZE>>>(rc->resolution().x*rc->resolution().y,
																			   rd->rays.device_memory,       rd->rays.tex,
																			   sd->vertex_pos.device_memory, sd->vertex_pos.tex,
																			   sd->triangles.device_memory,  sd->triangles.tex,
																			   rt->bvh_index.device_memory,  rt->bvh_index.tex,
																			   rt->bvh_nodes.device_memory,  rt->bvh_nodes.tex,
																			   rd->intersections.device_memory,
																			   anyhit);
			CHECK_CUDA_ERROR(hipGetLastError());
  			CHECK_CUDA_ERROR(hipDeviceSynchronize());
		}
		
		
		void persistentspeculativewhilewhile::compute_hit(bool anyhit) {
			auto *rt = dynamic_cast<wf::cuda::persistentspeculativewhilewhile*>(rc->scene.batch_rt);
			assert(rt != nullptr);

			persistentspeculativewhilewhile_trace<<<DESIRED_BLOCKS_COUNT, DESIRED_BLOCK_SIZE>>>(rc->resolution().x * rc->resolution().y,
																								rd->rays.device_memory,       rd->rays.tex,
																								sd->vertex_pos.device_memory, sd->vertex_pos.tex,
																								sd->triangles.device_memory,  sd->triangles.tex,
																								rt->bvh_index.device_memory,  rt->bvh_index.tex,
																								rt->bvh_nodes.device_memory,  rt->bvh_nodes.tex,
																								rd->intersections.device_memory,
																								anyhit);
			CHECK_CUDA_ERROR(hipGetLastError());
  			CHECK_CUDA_ERROR(hipDeviceSynchronize());
		}
		
		
		void persistentwhilewhile::compute_hit(bool anyhit) {
			auto *rt = dynamic_cast<wf::cuda::persistentwhilewhile*>(rc->scene.batch_rt);
			assert(rt != nullptr);

			persistentwhilewhile_trace<<<DESIRED_BLOCKS_COUNT, DESIRED_BLOCK_SIZE>>>(rc->resolution().x*rc->resolution().y,
																					 rd->rays.device_memory,       rd->rays.tex,
																					 sd->vertex_pos.device_memory, sd->vertex_pos.tex,
																					 sd->triangles.device_memory,  sd->triangles.tex,
																					 rt->bvh_index.device_memory,  rt->bvh_index.tex,
																					 rt->bvh_nodes.device_memory,  rt->bvh_nodes.tex,
																					 rd->intersections.device_memory,
																					 anyhit);
			CHECK_CUDA_ERROR(hipGetLastError());
  			CHECK_CUDA_ERROR(hipDeviceSynchronize());
		}

	}
}
