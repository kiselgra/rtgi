#include "hip/hip_runtime.h"
#include <optix_device.h>
#include "optix-launch-params.h"
#include <texture_indirect_functions.h>
#include <hip/hip_vector_types.h>
#include <vector_functions.h>
#include "cuda-operators.h"
#include "base.h"


namespace wf::cuda {

    extern "C" __constant__ optix_launch_params launch_params;
    
    static __forceinline__ __device__ void* unpack_pointer(uint32_t i0, uint32_t i1) {
        const uint64_t uptr = static_cast<uint64_t>(i0) << 32 | i1;
        void *ptr = reinterpret_cast<void*>(uptr);
        return ptr;
    };
    
    static __forceinline__ __device__ void pack_pointer(void *ptr, uint32_t &i0, uint32_t &i1) {
        const uint64_t uptr = reinterpret_cast<uint64_t>(ptr);
        i0 = uptr >> 32;
        i1 = uptr & 0x00000000FFFFFFFF;
    };

    template <typename T>
    static __forceinline__ __device__ T* per_ray_data() {
        const uint32_t u0 = optixGetPayload_0();
        const uint32_t u1 = optixGetPayload_1();
        return reinterpret_cast<T*>(unpack_pointer(u0, u1));
    };

    enum {SURFACE_RAY_TYPE = 0, RAY_TYPE_COUNT};

    extern "C" __global__ void __closesthit__radiance() {
        tri_is *prd = per_ray_data<tri_is>();

        prd->ref = optixGetPrimitiveIndex();
        const float2 barycentrics = optixGetTriangleBarycentrics();
        
        prd->beta = barycentrics.x;
        prd->gamma = barycentrics.y;
        prd->t = optixGetRayTmax();
    };

    extern "C" __global__ void __anyhit__radiance() {};
    extern "C" __global__ void __miss__radiance() {};
    
    /* \brief The raygen program does not generate any rays in this case.
     * Since our algorithm has a dedicated step for generating rays we store
     * a pointer to those in constant memory / the launch params and use them
     * within this function to call optixTrace.
     */
    extern "C" __global__ void __raygen__render_frame() {
        const int ix = optixGetLaunchIndex().x;
        const int iy = optixGetLaunchIndex().y;

        tri_is intersection;
        
        uint32_t u0, u1;
        pack_pointer(&intersection, u0, u1);
        
        int pixel_index = ix + iy * launch_params.frame_buffer_dimensions.x;
        
        float4 ray_o_f4 = launch_params.rays[pixel_index * 2];
        float4 ray_d_f4 = launch_params.rays[pixel_index * 2 + 1];
    
        float3 ray_origin_f3  = make_float3(ray_o_f4.x, ray_o_f4.y, ray_o_f4.z);
        float3 ray_direction_f3  = make_float3(ray_d_f4.x, ray_d_f4.y, ray_d_f4.z);
        
        optixTrace(launch_params.optix_traversable_handle,
                   ray_origin_f3,
                   ray_direction_f3,
                   0.f,
                   __FLT_MAX__,
                   0.0f,
                   OptixVisibilityMask(255),
                   launch_params.ray_flags,
                   SURFACE_RAY_TYPE,
                   RAY_TYPE_COUNT,
                   SURFACE_RAY_TYPE,
                   u0,
                   u1);
       
        launch_params.triangle_intersections[pixel_index] = intersection;
    }
}