#include "persistent-if-if.h"

#include "libgi/timer.h"

#include <iostream>

#include "kernels.h"
#include "cuda-helpers.h"

namespace wf{
	namespace cuda{
		void persistentifif::compute_hit(bool anyhit) {
			auto *rt = dynamic_cast<wf::cuda::persistentifif*>(rc->scene.batch_rt);
			assert(rt != nullptr);

			persistentififTrace<<<DESIRED_BLOCKS_COUNT, DESIRED_BLOCK_SIZE>>>(rc->resolution().x*rc->resolution().y,
															rd->rays.device_memory,
															rd->rays.tex,
															sd->vertex_pos.device_memory,
															sd->vertex_pos.tex,
															sd->triangles.device_memory,
															sd->triangles.tex,
															rt->bvh_index.device_memory,
															rt->bvh_index.tex,
															rt->bvh_nodes.device_memory,
															rt->bvh_nodes.tex,
															rd->intersections.device_memory,
															anyhit);
			CHECK_CUDA_ERROR(hipGetLastError());
  			CHECK_CUDA_ERROR(hipDeviceSynchronize());
		}
	}
}
