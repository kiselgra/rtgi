#include "dynamic-while-while.h"

#include "libgi/timer.h"

#include <iostream>

#include "kernels.h"
#include "cuda-helpers.h"

namespace wf{
	namespace cuda{
		void dynamicwhilewhile::compute_hit(bool anyhit) {
			auto *rt = dynamic_cast<wf::cuda::dynamicwhilewhile*>(rc->scene.batch_rt);
			assert(rt != nullptr);

			// dynamicwhilewhileTrace<<<DESIRED_BLOCKS_COUNT, DESIRED_BLOCK_SIZE>>>(
			// dynamicwhilewhile-Kernel uses 48 Registers instead of 40, so run one less warp than usual for best occupation
			dynamicwhilewhileTrace<<<DESIRED_BLOCKS_COUNT, dim3(WARPSIZE, DESIRED_WARPS_PER_BLOCK-1, 1)>>>(
														    rc->resolution().x * rc->resolution().y,
															rd->rays.device_memory,
															rd->rays.tex,
															sd->vertex_pos.device_memory,
															sd->vertex_pos.tex,
															sd->triangles.device_memory,
															sd->triangles.tex,
															rt->bvh_index.device_memory,
															rt->bvh_index.tex,
															rt->bvh_nodes.device_memory,
															rt->bvh_nodes.tex,
															rd->intersections.device_memory,
															anyhit);
			CHECK_CUDA_ERROR(hipGetLastError());
  			CHECK_CUDA_ERROR(hipDeviceSynchronize());
		}
	}
}
