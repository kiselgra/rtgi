#include "hip/hip_runtime.h"
#include "base.h"
#include "rni.h"
#include "tracers.h"

#include <iostream>

#define error(x) { std::cerr << "command (" << command << "): " << x << std::endl;  return true; }
#define check_in_complete(x) { if (in.bad() || in.fail() || !in.eof()) error(x); }

namespace wf {
	namespace cuda {

		platform::platform(const std::vector<std::string> &args) : wf::platform("cuda") {
			for (auto arg : args)
				std::cerr << "Platform opengl does not support the argument " << arg << std::endl;
			register_batch_rt("simple",, simple_rt);
			register_batch_rt("if-if",, ifif);
			register_batch_rt("while-while",, whilewhile);
			register_batch_rt("persistent-if-if",, persistentifif);
			register_batch_rt("persistent-while-while",, persistentwhilewhile);
			register_batch_rt("speculative-while-while",, speculativewhilewhile);
			register_batch_rt("persistent-speculative-while-while",, persistentspeculativewhilewhile);
			register_batch_rt("dynamic-while-while",, dynamicwhilewhile);

			link_tracer("while-while", "default");
			link_tracer("while-while", "find closest hits");
			// bvh mode?
			register_rni_step_by_id(, initialize_framebuffer);
			register_rni_step_by_id(, batch_cam_ray_setup);
			//register_rni_step("store hitpoint albedo",, store_hitpoint_albedo_cpu);
			register_rni_step_by_id(, add_hitpoint_albedo_to_fb);
			register_rni_step_by_id(, download_framebuffer);
		}

		platform::~platform() {
			hipDeviceReset();
		}

		void scenedata::upload(scene *scene) {
			std::vector<uint4> scene_tris;
			scene_tris.reserve(scene->triangles.size());
			for (triangle t : scene->triangles)
				scene_tris.push_back(uint4{t.a, t.b, t.c, t.material_id});
			triangles.upload(scene_tris.size(), reinterpret_cast<uint4*>(scene_tris.data()));

			int num_vertices = scene->vertices.size();
			std::vector<float4> tmp4(num_vertices);
			std::vector<float2> tmp2(num_vertices);

			for (int i = 0; i < num_vertices; ++i) {
				tmp4[i] = float4{ scene->vertices[i].pos.x, scene->vertices[i].pos.y, scene->vertices[i].pos.z, 0 };
				tmp2[i] = float2{ scene->vertices[i].tc.x, scene->vertices[i].tc.y };
			}
			vertex_pos.upload(tmp4);
			vertex_tc.upload(tmp2);

			auto f4 = [](const vec3 &v) { return float4{ v.x, v.y, v.z, 0 }; };
			std::vector<material> mtls(scene->materials.size());
			for (int i = 0; i < scene->materials.size(); ++i) {
				mtls[i].albedo = f4(scene->materials[i].albedo);
				mtls[i].emissive = f4(scene->materials[i].emissive);
				if (scene->materials[i].albedo_tex) {
					texture_image ti(*scene->materials[i].albedo_tex);
					tex_images.push_back(ti);
					mtls[i].albedo_tex = ti.tex;
				}
				else
					mtls[i].albedo_tex = 0;
			}
			materials.upload(mtls);
		}

		void batch_rt::build(::scene *scene)
		{
			rd = new raydata(rc->resolution());
			sd = new scenedata;

			binary_bvh_tracer<bbvh_triangle_layout::indexed, bbvh_esc_mode::on> bvh_rt;
			if (bvh_type == "sah")     bvh_rt.binary_split_type = bvh_rt.sah;
			else if (bvh_type == "sm") bvh_rt.binary_split_type = bvh_rt.sm;
			else if (bvh_type == "om") bvh_rt.binary_split_type = bvh_rt.om;
			bvh_rt.max_triangles_per_node = bvh_max_tris_per_node;
			bvh_rt.build(scene);

			// bvh_index.upload(bvh_rt.index);
			std::vector<uint1> new_index_list;
			for (auto index : bvh_rt.index) {
				uint1 new_index;
				new_index.x = index;
				new_index_list.push_back(new_index);
			}
			bvh_index.upload(new_index_list);

			bvh_nodes.upload(compact_bvh_node_builder::build(bvh_rt.nodes));

			auto *rt = dynamic_cast<batch_rt*>(rc->scene.batch_rt);
			assert(rt != nullptr);
			sd->upload(scene);
			std::cout << "upload done" << std::endl;
		}

		bool batch_rt::interprete(const std::string &command, std::istringstream &in) {
			if (command == "incoherence") {
				float in_r1, in_r2;
				in >> in_r1;
				in >> in_r2;
				check_in_complete("Syntax error, \"incoherence\" requires exactly two positive float values");
				if (in_r1 < 0 || in_r2 < 0)
					error("Parameter error, \"incoherence\" requires exactly two positive float values");
				if (in_r1 != 0 || in_r2 != 0) {
					use_incoherence = true;
					incoherence_r1 = in_r1;
					incoherence_r2 = in_r2;
				}
				return true;
			}
			else if (command == "bvh") {
				std::string sub;
				in >> sub;
				if (sub == "type") {
					std::string in1;
					in >> in1;
					check_in_complete("Syntax error, \"bvh type\" requires exactly one string value");
					bvh_type = in1;
					if (in1 != "sah" && in1 != "sm" && in1 != "om")
						error("Parameter error, \"bvh type\" must be one of \"sm\", \"om\", \"sah\"");
					return true;
				}
				else if (sub == "max_tris") {
					int in1;
					in >> in1;
					check_in_complete("Syntax error, \"bvh max_tris\" requires exactly one positive integer value");
					bvh_max_tris_per_node = in1;
					return true;
				}
			}
			return false;
		}

		__host__ std::vector<compact_bvh_node> compact_bvh_node_builder::build(std::vector<binary_bvh_tracer<bbvh_triangle_layout::indexed, bbvh_esc_mode::on>::node> nodes) {
			std::vector<wf::cuda::compact_bvh_node> nodes_new;
			for (const auto& n : nodes) {
				wf::cuda::compact_bvh_node node;
				node.data1 = make_float4(n.box_l.min.x, n.box_l.max.x, n.box_l.min.y, n.box_l.max.y);
				node.data2 = make_float4(n.box_r.min.x, n.box_r.max.x, n.box_r.min.y, n.box_r.max.y);
				node.data3 = make_float4(n.box_l.min.z, n.box_l.max.z, n.box_r.min.z, n.box_r.max.z);

				// change links on inner nodes to indicate wether child is inner node or leaf node
				if (n.inner()) {
					*(int*)&node.data4.x = nodes[n.link_l].inner() ? n.link_l : -n.link_l;
					*(int*)&node.data4.y = nodes[n.link_r].inner() ? n.link_r : -n.link_r;
				}
				else {
					*(int*)&node.data4.x = n.link_l;	// tri_offset
					*(int*)&node.data4.y = n.link_r;	// tri_count
				}
				nodes_new.push_back(node);
			}
			assert(nodes_new.size() == nodes.size());
			return nodes_new;
		}

	}
}
